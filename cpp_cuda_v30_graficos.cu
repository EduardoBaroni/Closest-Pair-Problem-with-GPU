#include "hip/hip_runtime.h"
/* 
	Este programa trata do problema de se achar os pares de pontos mais próximos num plano xy. 
	Aqui se utiliza a plataforma CUDA a fim de verificar possíveis soluções eficientes.
*/

// Bibliotecas C
#include <stdio.h>
#include <stdlib.h> 
#include <math.h>
#include <limits.h>
#include <time.h>

// Bibliotecas Thrust
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/extrema.h>

// Bibliotecas C++
#include <iostream>
#include <fstream>
#include <iterator>

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/
void geraDados(float y, int x)
{
	FILE *y_data = fopen("eixoVertical_cuda", "a");	
	fprintf(y_data, "%g", y / (float) CLOCKS_PER_SEC);
	fprintf(y_data, "%s", "\n");
	
	FILE *x_data = fopen("eixoHorizontal_cuda", "a");	
	fprintf(x_data, "%d", x);
	fprintf(x_data, "%s", "\n");

	fclose(y_data);
	fclose(x_data);
}

// Funções
//PRESTE ATENÇÃO NO '&', significa que se esta passando vector por referência
void leitura(char *argv[], unsigned int *num_pontos, thrust::host_vector<int> &hX, thrust::host_vector<int> &hY)

{
	std::ifstream pts(argv[1], std::ios::binary);
	std::ifstream file(argv[2], std::ios::binary);


	if(file && pts)// Só verificando se não deu falha ao abrir os arquivos
	{
		// Inicializando num_pontos
		pts.read((char*) num_pontos, sizeof(int));

 		// Após obter num pontos pode-se alocar dinamicamente os host vectors
 		hX.resize(*num_pontos);
 		hY.resize(*num_pontos);	
 		
 		// Entao as coordenadas são lidas
		file.read((char*)(hX.data()), hX.size()*sizeof(int));
		file.read((char*)(hY.data()), hY.size()*sizeof(int));
	}

	pts.close();
	file.close();
}

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

int calculaRegioes(unsigned int num_pontos, unsigned int ptsRegiao)
{
	int num_regioes;

	num_regioes = num_pontos / ptsRegiao;	
	
	if( num_pontos % ptsRegiao != 0 )
		num_regioes ++;
	
	return num_regioes;
}

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

// Kernels

__global__ void calculaDistancias(unsigned int num_pontos, int *X, int *Y, float *dD)
{
	int idg = blockIdx.x * blockDim.x + threadIdx.x; // Índice global da thread corrente
	int xi, xii, yi, yii ;

	long int A,B;

	if( idg < num_pontos-1 ){

		xi  = X[idg];
		xii = X[idg+1];
		yi  = Y[idg];
		yii = Y[idg+1];

		if( xi!=xii || yi!=yii ){

			A = (long int) ( (long int)(xi - xii) * (long int)(xi - xii) );
								
			B = (long int) ( (long int)(yi - yii) * (long int)(yi - yii) );
		
			dD[idg] =  (float) sqrt( (double) (A + B) );
		}
		else{

			dD[idg] = INT_MAX;
		}
	}
}

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

__global__ void Forca_Bruta(int num_pontos, int num_regioes, int ptsRegiao, int *X, int *Y, float *Minimos, float delta_inicial)
{

	int idb = blockIdx.x; // Índice do bloco corrente (coincide com a região corrente)
	int idg = blockIdx.x * blockDim.x + threadIdx.x; // Índice global da thread corrente
	int k; // auxiliar
	float aux, delta_minimo = delta_inicial;
	long int A,B;
	int LimFinal, x_final;
	int xi, xk, yi, yk ;

	xi = X[idg];
	yi = Y[idg];

	if( idb < num_regioes-1 )// Todos as regioes menos a última são tratadas igualmente.
	{ 
		// Calculo do limite final da região
		x_final = X[ptsRegiao*(idb+1)-1];
		LimFinal = x_final + (int) delta_inicial;

		for( k=idg+1 ; X[k]<=LimFinal && k<num_pontos ; k++ ){ // cada thread executará esse laço.

			xk = X[k];
			yk = Y[k];

			// OTIMIZAÇÃO: Olhar a coordenada x
			if(xk-xi>(int)delta_minimo ){
				k = num_pontos;
			}
			else if( xi!=xk || yi!=yk ){

				A = (long int) ( (long int)(xi-xk)*(long int)(xi-xk) );
			
				B = (long int) ( (long int)(yi-yk)*(long int)(yi-yk) );
	
				aux = (float) sqrt( (double) (A + B) );

				if( aux < delta_minimo ){
					delta_minimo = aux;
					LimFinal = x_final + (int) delta_minimo;
				}
			}
		}
		Minimos[idg] = delta_minimo;
	}
	else
	{
		if( idg < num_pontos-1 ){

			for( k=idg+1 ; k < num_pontos ; k++ ){ // cada thread executará esse laço.

				xk = X[k];
				yk = Y[k];

				// OTIMIZAÇÃO: Olhar a coordenada x
				if(xk-xi>(int)delta_minimo ){
					k = num_pontos;
				}
				else if( xi!=xk || yi!=yk ){

					A = (long int) ( (long int)(xi-xk)*(long int)(xi-xk) );
				
					B = (long int) ( (long int)(yi-yk)*(long int)(yi-yk) );
		
					aux = (float) sqrt( (double) (A + B) );

					if( aux < delta_minimo )
						delta_minimo = aux;
				}
			}
			Minimos[idg] = delta_minimo;
		}
	}
}

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

int main(int argc, char *argv[])
{
	clock_t inicio = clock();
	// Declaração de variáveis:
	unsigned int num_regioes, num_pontos, ptsRegiao;
	int maxThreadBloco;
	float delta_inicial, delta_minimo;

	// Capturando o máximo número de threads por bloco da máquina
	hipDeviceGetAttribute(&maxThreadBloco, hipDeviceAttributeMaxThreadsPerBlock,0);
	ptsRegiao = maxThreadBloco/32;

	// HOST
	thrust::host_vector<int> hX; // Coordenadas x no host
	thrust::host_vector<int> hY; // Coordenadas y no host

	// DEVICE
	thrust::device_vector<int> dX; // Coordenadas x no device
	thrust::device_vector<int> dY; // Coordenadas y no device

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

	// Passo 1: Leitura e armazenamento dos pontos. Esse passo é feito lendo um arquivo binário.

	clock_t inicio_leitura = clock();
	leitura(argv, &num_pontos, hX, hY);
	clock_t fim_leitura = clock();

	printf("\nTempo da função leitura: %g segundos\n\n", (fim_leitura - inicio_leitura) / (float) CLOCKS_PER_SEC);

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

	// Passo 2: Memcpy's do host para device

	clock_t inicio_transferencia = clock();
	dX = hX;
	dY = hY;
	clock_t fim_transferencia = clock();

	printf("Tempo da transferencia: %g segundos\n\n", (fim_transferencia - inicio_transferencia) / (float) CLOCKS_PER_SEC);

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

	// Passo 3: Ordenando os pontos em X:

	clock_t inicio_ordenacao = clock();
	thrust::stable_sort_by_key(dX.begin(), dX.end(), dY.begin());
	clock_t fim_ordenacao = clock();

	printf("Tempo da função de ordenação: %g segundos\n\n", (fim_ordenacao - inicio_ordenacao) / (float) CLOCKS_PER_SEC);

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

	// Passo 4: Dividir os n pontos que temos em m regioes, de forma que cada bloco tenha aproximadamente a mesma quantidade de pontos.

	num_regioes = calculaRegioes(num_pontos, ptsRegiao);

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

	// Passo 5: Calculando o delta inicial (distância euclidiana mínima entre um ponto e seu sucessor armazenado):

	// INICIO MEDIÇÃO DE TEMPO:
	clock_t inicio_calc_distancias = clock();

	thrust::device_vector<float> dD(num_pontos-1); // Vetor de Distâncias (para o delta inicial) no device	
	
	// Forma encontrada de usar vetores da thrust em um kernel: Apontar para cada um deles com novos ponteiros.
	int *X = thrust::raw_pointer_cast(&dX[0]); // aponta para dX
	int *Y = thrust::raw_pointer_cast(&dY[0]); // aponta para dY
	float *d = thrust::raw_pointer_cast(&dD[0]); // aponta para dD

	//Número Máximo de Blocos: 2^31-1 = 2 147 483 647
	int num_blocos;
	
	if( num_pontos % maxThreadBloco != 0 )
		num_blocos = (num_pontos / maxThreadBloco) + 1;
	else
		num_blocos = num_pontos / maxThreadBloco;		

 	// Kernel que calcula vector de distâncias
	calculaDistancias<<<num_blocos, maxThreadBloco>>>(num_pontos, X, Y, d);
	
 	hipDeviceSynchronize(); // Necessário

	// FIM MEDIÇÃO DE TEMPO
	clock_t fim_calc_distancias = clock();

	printf("Tempo do kernel Calcula Distâncias: %g segundos\n\n", (fim_calc_distancias - inicio_calc_distancias) / (float) CLOCKS_PER_SEC);

	// Redução usando thrust para achar delta inicial do vetor de distâncias
	clock_t inicio_reducao1 = clock();
	thrust::device_vector<float>::iterator iter = thrust::min_element(dD.begin(), dD.end()); 

	delta_inicial = *iter;
	printf("\n\nDelta Inicial: %lf\n\n", delta_inicial);

	clock_t fim_reducao1 = clock();

	printf("Tempo da redução1: %g segundos\n\n", (fim_reducao1 - inicio_reducao1) / (float) CLOCKS_PER_SEC);

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

	// Passo 6: Para cada bloco, achar seu delta, utilizando algoritmo de força bruta.

	// INICIO MEDIÇÃO DE TEMPO
	clock_t inicio_forca_bruta = clock();

	if( num_regioes%maxThreadBloco != 0 )
		num_blocos = (num_regioes/maxThreadBloco) + 1;
	else
		num_blocos = num_regioes/maxThreadBloco;	

	thrust::device_vector<float> dMin(num_pontos, INT_MAX); // Vetor de minimos
	
	float *Min = thrust::raw_pointer_cast(&dMin[0]); // aponta para dMin 

	Forca_Bruta<<<num_regioes, ptsRegiao>>>(num_pontos, num_regioes, ptsRegiao, X, Y, Min, delta_inicial);
	
	hipDeviceSynchronize();

	// FIM MEDICAO DE TEMPO
	clock_t fim_forca_bruta = clock();

	printf("Tempo do kernel Força Bruta: %g segundos\n\n", (fim_forca_bruta - inicio_forca_bruta) / (float) CLOCKS_PER_SEC);
	
	// Redução do vetor dMin:
	clock_t inicio_reducao2 = clock();
	thrust::device_vector<float>::iterator iter2 = thrust::min_element(dMin.begin(), dMin.end());
	
	delta_minimo = *iter2;

	// Imprimindo resultados:
	printf("Delta mínimo:\n%lf\n", delta_minimo);
	clock_t fim_reducao2 = clock();

	printf("Tempo da redução2: %g segundos\n\n", (fim_reducao2 - inicio_reducao2) / (float) CLOCKS_PER_SEC);

	clock_t fim = clock();
	printf("Tempo total: %g segundos\n\n", (fim - inicio) / (float) CLOCKS_PER_SEC);

	geraDados(fim-inicio, num_pontos);
	
	return 0;
}
