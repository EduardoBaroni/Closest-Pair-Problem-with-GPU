#include "hip/hip_runtime.h"
/* 
	Este programa trata do problema de se achar os pares de pontos mais próximos num plano xy. 
	Aqui se utiliza a plataforma CUDA a fim de verificar possíveis soluções eficientes.
*/

// Bibliotecas C
#include <stdio.h>
#include <stdlib.h> 
#include <math.h>
#include <limits.h>
#include <time.h>

// Bibliotecas Thrust
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/extrema.h>

// Bibliotecas C++
#include <iostream>
#include <fstream>
#include <iterator>

// Bibliotecas Nvprof
#include "hip/hip_runtime_api.h"

// Kernels

__global__ void calculaDistancias(unsigned int num_pontos, int *X, int *Y, float *dD)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x; // thread corrente

	long int A,B;

	if( index < num_pontos-1 ){


		if( X[index]!=X[index+1] || Y[index]!=Y[index+1] ){

			A = (long int) ( (long int)(X[index] - X[index+1]) * (long int)(X[index] - X[index+1]) );
								
			B = (long int) ( (long int)(Y[index] - Y[index+1]) * (long int)(Y[index] - Y[index+1]) );
		
			dD[index] =  (float) sqrt( (double) (A + B) );

		}
		else{

			dD[index] = INT_MAX;

		}

	}

}

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

__global__ void Forca_Bruta(int num_pontos, int num_regioes, int ptsRegiao, int *X, int *Y, float *Minimos, float delta_inicial)
{

	int i = blockIdx.x; // bloco corrente (coincide com a região corrente)
	int j = blockIdx.x * blockDim.x + threadIdx.x; // thread corrente
	int k; // auxiliar
	float aux, delta_minimo = delta_inicial;
	long int A,B;
	int LimFinal;

	// Passo 5:
	if( i < num_regioes-1 )// Todos as regioes menos a última são tratadas igualmente.
	{ 
		// Calculo do limite final da região
		LimFinal = X[ptsRegiao * (i+1) - 1] + (int) delta_inicial;

		for( k=j+1 ; X[k]<=LimFinal && k<num_pontos ; k++ ){ // cada thread executará esse laço.

			if( X[j] != X[j+1] || Y[j] != Y[j+1] ){

				A = (long int) ( (long int)(X[j] - X[k]) * (long int)(X[j] - X[k]) );
								
				B = (long int) ( (long int)(Y[j] - Y[k]) * (long int)(Y[j] - Y[k]) );
		
				aux =  (float) sqrt( (double) (A + B) );

				if( aux < delta_minimo )
					delta_minimo = aux;
			}
		}
		Minimos[j] = delta_minimo;
	}
	else
	{
		if( j < num_pontos-1 ){

			for( k=j+1 ;  k < num_pontos ; k++ ){ // cada thread executará esse laço.

				if( X[j] != X[j+1] || Y[j] != Y[j+1] ){

					A = (long int) ( (long int)(X[j] - X[k]) * (long int)(X[j] - X[k]) );
								
					B = (long int) ( (long int)(Y[j] - Y[k]) * (long int)(Y[j] - Y[k]) );
		
					aux =  (float) sqrt( (double) (A + B) );

					if( aux < delta_minimo )
						delta_minimo = aux;
				}
			}
			Minimos[j] = delta_minimo;
		}
	}
}

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

// Funções
//PRESTE ATENÇÃO NO '&', significa que se esta passando vector por referência
void leitura(char *argv[], unsigned int *num_pontos, thrust::host_vector<int> &hX, thrust::host_vector<int> &hY)

{
	std::ifstream pts(argv[1], std::ios::binary);
	std::ifstream file(argv[2], std::ios::binary);


	if(file && pts)// Só verificando se não deu falha ao abrir os arquivos
	{
		// Inicializando num_pontos
		pts.read((char*) num_pontos, sizeof(int));

 		// Após obter num pontos pode-se alocar dinamicamente os host vectors
 		hX.resize(*num_pontos);
 		hY.resize(*num_pontos);	
 		
 		// Entao as coordenadas são lidas
		file.read((char*)(hX.data()), hX.size()*sizeof(int));
		file.read((char*)(hY.data()), hY.size()*sizeof(int));

	}

	pts.close();
	file.close();
}

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

int calculaRegioes(unsigned int num_pontos, unsigned int ptsRegiao)
{
	int num_regioes;

	num_regioes = num_pontos / ptsRegiao;	
	
	if( num_pontos % ptsRegiao != 0 )
		num_regioes += 1;
	
	return num_regioes;
}

inline void cpp_medicoes(char* argv[]);

/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

int main(int argc, char *argv[])
{
	clock_t inicio = clock();
	
/*-----------------------------------------------------------------------------------------------------------------*/
/*-----------------------------------------------------------------------------------------------------------------*/

	#if MEDICAO_INTERMEDIARIA
		cpp_medicoes(argv);
		
	#else	
		// Declaração de variáveis:
		unsigned int num_regioes, num_pontos, ptsRegiao;
		int maxThreadBloco;
		float delta_inicial, delta_minimo;

		// Capturando o máximo número de threads por bloco da máquina
		hipDeviceGetAttribute(&maxThreadBloco, hipDeviceAttributeMaxThreadsPerBlock,0);
		ptsRegiao = maxThreadBloco/32;

		// HOST
		thrust::host_vector<int> hX; // Coordenadas x no host
		thrust::host_vector<int> hY; // Coordenadas y no host

		// DEVICE
		thrust::device_vector<int> dX; // Coordenadas x no device
		thrust::device_vector<int> dY; // Coordenadas y no device
		
		// Passo 1: Leitura e armazenamento dos pontos. Esse passo é feito lendo um arquivo binário.

	 	leitura(argv, &num_pontos, hX, hY);

		// Passo 2: Memcpy's do host para device

		dX = hX;
		dY = hY;
		
		// Passo 3: Ordenando os pontos em X:

		thrust::stable_sort_by_key(dX.begin(), dX.end(), dY.begin());

		// Passo 4: Dividir os n pontos que temos em m regioes, de forma que cada bloco tenha aproximadamente a mesma quantidade de pontos.

		num_regioes = calculaRegioes(num_pontos, ptsRegiao);

		//Passo 5: Calculando o delta inicial (distância euclidiana mínima entre um ponto e seu sucessor armazenado):

		thrust::device_vector<float> dD(num_pontos-1); // Vetor de Distâncias (para o delta inicial) no device	
		
		// Forma encontrada de usar vetores da thrust em um kernel: Apontar para cada um deles com novos ponteiros.
		int *X = thrust::raw_pointer_cast(&dX[0]); // aponta para dX
		int *Y = thrust::raw_pointer_cast(&dY[0]); // aponta para dY
		float *d = thrust::raw_pointer_cast(&dD[0]); // aponta para dD

		//Número Máximo de Blocos: 2^31-1 = 2 147 483 647
		int num_blocos;
		
		if( num_pontos % maxThreadBloco != 0 )
			num_blocos = (num_pontos / maxThreadBloco) + 1;
		else
			num_blocos = num_pontos / maxThreadBloco;		

	 	// Kernel que calcula vector de distâncias
		calculaDistancias<<<num_blocos, maxThreadBloco>>>(num_pontos, X, Y, d);
		
	 	hipDeviceSynchronize();

		thrust::device_vector<float>::iterator iter = thrust::min_element(dD.begin(), dD.end()); // Redução usando thrust para achar delta inicial do vetor de distâncias
		
		delta_inicial = *iter;
		printf("\n\nDelta Inicial: %lf\n\n", delta_inicial);
		
		//Passo 6: Para cada bloco, achar seu delta, utilizando algoritmo de força bruta.

		if( num_regioes%maxThreadBloco != 0 )
			num_blocos = (num_regioes/maxThreadBloco) + 1;
		else
			num_blocos = num_regioes/maxThreadBloco;	

		thrust::device_vector<float> dMin(num_pontos, INT_MAX); // Vetor de minimos
		
		float *Min = thrust::raw_pointer_cast(&dMin[0]); // aponta para dMin 

		Forca_Bruta<<<num_regioes, ptsRegiao>>>(num_pontos, num_regioes, ptsRegiao, X, Y, Min, delta_inicial);
		
		hipDeviceSynchronize();

		// Redução do vetor dMin:
		iter = thrust::min_element(dMin.begin(), dMin.end());
		
		delta_minimo = *iter;
		printf("Delta mínimo:\n%lf\n", delta_minimo);
	#endif

	clock_t fim = clock();
	printf("Tempo total: %g segundos\n\n", (fim - inicio) / (float) CLOCKS_PER_SEC);

	return 0;
}


inline void cpp_medicoes(char* argv[])
{
	unsigned int num_pontos;
	thrust::host_vector<int> hX;
	thrust::host_vector<int> hY;

	clock_t inicio_leitura = clock();
 	leitura(argv, &num_pontos, hX, hY);
	clock_t fim_leitura = clock();

	printf("\nTempo da função leitura: %g segundos\n\n", (fim_leitura - inicio_leitura) / (float) CLOCKS_PER_SEC);

	clock_t inicio_transferencia = clock();
	thrust::device_vector<int> dX = hX;
	thrust::device_vector<int> dY = hY;
	clock_t fim_transferencia = clock();

	printf("Tempo da transferencia: %g segundos\n\n", (fim_transferencia - inicio_transferencia) / (float) CLOCKS_PER_SEC);

	clock_t inicio_ordenacao = clock();
	thrust::stable_sort_by_key(dX.begin(), dX.end(), dY.begin());
	clock_t fim_ordenacao = clock();

	printf("Tempo da função de ordenação: %g segundos\n\n", (fim_ordenacao - inicio_ordenacao) / (float) CLOCKS_PER_SEC);

	int maxThreadBloco;
	hipDeviceGetAttribute(&maxThreadBloco, hipDeviceAttributeMaxThreadsPerBlock,0);
	unsigned int ptsRegiao = maxThreadBloco/32;
	
	unsigned int num_regioes = calculaRegioes(num_pontos, ptsRegiao);

	thrust::device_vector<float> dD(num_pontos-1);
	
	int *X = thrust::raw_pointer_cast(&dX[0]);
	int *Y = thrust::raw_pointer_cast(&dY[0]);
	float *d = thrust::raw_pointer_cast(&dD[0]);

	int num_blocos = num_pontos % maxThreadBloco != 0 ? (num_pontos / maxThreadBloco) + 1 : num_blocos = num_pontos / maxThreadBloco;
	
	clock_t inicio_calc_distancias = clock();
	calculaDistancias<<<num_blocos, maxThreadBloco>>>(num_pontos, X, Y, d);
	
 	hipDeviceSynchronize();
	clock_t fim_calc_distancias = clock();

	printf("Tempo do kernel Calcula Distâncias: %g segundos\n\n", (fim_calc_distancias - inicio_calc_distancias) / (float) CLOCKS_PER_SEC);

	clock_t inicio_reducao1 = clock();
	thrust::device_vector<float>::iterator iter = thrust::min_element(dD.begin(), dD.end());
	clock_t fim_reducao1 = clock();

	float delta_inicial = *iter;

	printf("\n\nDelta Inicial: %lf\n\n", delta_inicial);
	printf("Tempo da redução1: %g segundos\n\n", (fim_reducao1 - inicio_reducao1) / (float) CLOCKS_PER_SEC);

	num_blocos = num_regioes % maxThreadBloco != 0 ? (num_regioes / maxThreadBloco) + 1 : num_blocos = num_regioes / maxThreadBloco;

	thrust::device_vector<float> dMin(num_pontos, INT_MAX);
	float *Min = thrust::raw_pointer_cast(&dMin[0]);

	clock_t inicio_forca_bruta = clock();
	Forca_Bruta<<<num_regioes, ptsRegiao>>>(num_pontos, num_regioes, ptsRegiao, X, Y, Min, delta_inicial);
	
	hipDeviceSynchronize();
	clock_t fim_forca_bruta = clock();

	printf("Tempo do kernel Força Bruta: %g segundos\n\n", (fim_forca_bruta - inicio_forca_bruta) / (float) CLOCKS_PER_SEC);
	
	clock_t inicio_reducao2 = clock();
	iter = thrust::min_element(dMin.begin(), dMin.end());
	clock_t fim_reducao2 = clock();

	float delta_minimo = *iter;

	printf("Delta mínimo:\n%lf\n", delta_minimo);
	printf("Tempo da redução2: %g segundos\n\n", (fim_reducao2 - inicio_reducao2) / (float) CLOCKS_PER_SEC);
}